#include "hip/hip_runtime.h"
#include "common.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int matrix_size;
int cluster_size;
float *matrix, *gpuRef, *hostRef;
float *dev_matrix;
float dev_det;

void DeviceFunc(float *temp_h , int numvar , float *temp1_h){
    float *a_d , *b_d;
    //Memory allocation on the device
    hipMalloc(&a_d,sizeof(float)*(numvar)*(numvar+1));
    hipMalloc(&b_d,sizeof(float)*(numvar)*(numvar+1));
    //Copying data to device from host
    hipMemcpy(a_d, temp_h, sizeof(float)*numvar*(numvar+1),hipMemcpyHostToDevice);
    //Defining size of Thread Block
    dim3 dimBlock(numvar+1,numvar,1);
    dim3 dimGrid(1,1,1);
    //Kernel call 
    //Kernel<<<dimGrid , dimBlock>>>(a_d , b_d , numvar);
    //Coping data to host from device
    hipMemcpy(temp1_h,b_d,sizeof(float)*numvar*(numvar+1),hipMemcpyDeviceToHost);
    //Deallocating memory on the device
    hipFree(a_d);
    hipFree(b_d);
}

__global__ void detMatrixOnGPUMix(float *matrix, int nx, int ny){
    int matrix_size, matrix_number, current_matrix;
    float deter, pivot, *line;

    matrix_size = nx;
    matrix_number = ny;
    current_matrix = blockIdx.y;

    unsigned int idxCollumn = threadIdx.x;
    unsigned int idxLine = threadIdx.y;
    unsigned int idxPosition = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idxPivot = current_matrix + idxCollumn * matrix_size + idxCollumn;

    pivot = matrix[0];
    printf("%d %d %d %d %f\n", current_matrix, idxCollumn, idxLine, idxPivot, pivot);
    //printf("idxCurrentMatrix: %d current_matrix: %d\n", idxCurrentMatrix, current_matrix);

    //for (int i = idxCurrentMatrix; i < (idxCurrentMatrix+1) * matrix_size * matrix_size; i += matrix_size) {
        //if (current_matrix == 0)
            //printf("%d %d %d %d\n", current_matrix, i, idxCollumn, i * matrix_size + idxCollumn);
    //}

    if(idxCollumn < matrix_size && idxLine < matrix_size){
	    /* Pivot Verification */
	    if(pivot == 0.000000){
			// Procurar novo pivot, diferente de 0
			int i = idxPivot;
			bool newpivot_found = false;
		        while(!newpivot_found){
				printf("A procurar novo pivot.\n");
				i = i + matrix_size;
				if(matrix[i] != 0){
					printf("Encontrou novo pivot.\n");
					newpivot_found = true;
					double aux = (i-1) / matrix_size;
					i = floor(aux) * matrix_size + 1;
					// Guardar valores da linha num novo array. E trocar valores entre linhas
					for(int k = idxPosition; k < idxPosition + matrix_size; k++) {
						line[k-1] = matrix[k];
						matrix[k] = matrix[i];
						matrix[i] = line[k-1];
						i++;
					}
				}
			}
	    }

	    // thread N-1 calcula determinante no final
	    if((threadIdx.x == nx-1) && (threadIdx.y == nx-1)) {
		    /* Determinant Calculation */
		    deter = 1;
		    //printf("A calcular determinante.\n");
		    for(int i = 0; i < matrix_size; i++)
			    deter *= matrix[i*i];

		    printf("Matrix number - %d; Determinante - %d.\n", current_matrix, deter);
		    
	    }// restantes threads preenchem colunas a 0
	    else {
		    //printf("A aplicar eliminacao de Gauss.\n");
		    /* Gauss Elimination */
		//printf("%d %d\n", matrix_size - idxLine+1, matrix_size - idxCollumn+1);
		for(int i = idxCollumn; i < matrix_size; i++) {
		    for(int k = idxLine; k < matrix_size; k++) {
			   	matrix[(k*matrix_size)+(i+1)] = matrix[i+1] * matrix[k*matrix_size] - matrix[(k*matrix_size)+(i+1)] * matrix[0];
		   	 }
		}
		    //__syncthreads();
	    }
	}	    
}

int main(int argc, char **argv){
    char *fName;
	
    if (argc != 2){
        printf("Usage: ./<ExecutableFilename> <MatrixFilename>\n");
	printf("%6s MatrixFilename: Name the file that contains the matrices\n", "->");
	exit(1);
    }
	
    fName = argv[1];
    
    printf("%s \nStarting...\n", argv[0]);

    int matrix_number, current_matrix, matrixreceived = 0;
    FILE *matrix_file = fopen(fName, "rb");
	
    if(matrix_file == NULL) {
        printf("Cannot open the %s file\n"
               "Please check if is a valid matrix file and the user read permissions!\n", argv[1]);
        return 1;
    }
	
    fread(&matrix_number, sizeof(int), 1, matrix_file);
    fread(&matrix_size, sizeof(int), 1, matrix_file);
    int dimension  = matrix_number * matrix_size;

    printf("Matrix size: %d | Matrix count: %d\n", matrix_size, matrix_number);
    
    //Read the matrices on file 
    matrix = (float*) malloc(sizeof(float) * (matrix_size * matrix_size * matrix_number));
    for(current_matrix = 0; current_matrix < matrix_number; current_matrix++){
        for(int i = current_matrix * matrix_size * matrix_size; i < (current_matrix + 1) * matrix_size * matrix_size; i++) {
            fread(&matrix[i], sizeof(float), 1, matrix_file);
            //printf("%d %f\n", i, matrix[i]);
        }
    }
    
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = matrix_size;
    int ny = matrix_number;

    int nxy = nx * nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    //hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    double iStart = seconds();

    double iElaps = seconds() - iStart;
    printf("Matrix initialization elapsed %f sec\n", iElaps);

    //memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    iStart = seconds();

    //Multiply Matrix OnHost(h_A, h_B, hostRef, nx, ny);
    //multMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = seconds() - iStart;
    
    //printf("multMatrixOnHost elapsed %f sec\n", iElaps);

    CHECK(hipMalloc((void **)&dev_matrix, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(dev_matrix, matrix, nBytes, hipMemcpyHostToDevice));
    
    dim3 grid(1,ny);
    dim3 block(nx,nx);

    iStart = seconds();
    detMatrixOnGPUMix<<<grid, block>>>(dev_matrix, nx, ny);
    
    //CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("detMatrixOnGPUMix <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x, grid.y, block.x, block.y, iElaps);

    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    //CHECK(hipMemcpy(gpuRef, dev_matrix, nBytes, hipMemcpyDeviceToHost));

    // check device results
    //checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(dev_matrix));

    // free host memory
    free(matrix);
    //free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}

#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

 int matrix_size;
 int cluster_size;
 float *matrix, *gpuRef, *hostRef;
 float *dev_matrix;
 float dev_det;
 
/* __device__ __global__ void Kernel(float *a_d , float *b_d , int size){

    int idx = threadIdx.x;
    int idy = threadIdx.y;

    //Allocating memory in the share memory of the device
    __shared__ float temp[16][16];

    //Copying the data to the shared memory
    temp[idy][idx] = a_d[(idy * (size+1)) + idx]  ;
    for(int i =1 ; i<size ;i++){
        if((idy + i) < size)// NO Thread divergence here
        {
            float var1 =(-1)*( temp[i-1][i-1]/temp[i+idy][i-1]);
            temp[i+idy][idx] = temp[i-1][idx] +((var1) * (temp[i+idy][idx]));
        }
        __syncthreads();  //Synchronizing all threads before Next iteration
    } 
    b_d[idy*(size+1) + idx] = temp[idy][idx];
}
*/
//__device__ __global__ void Kernel(float *, float * ,int );

void DeviceFunc(float *temp_h , int numvar , float *temp1_h){
    float *a_d , *b_d;
    //Memory allocation on the device
    hipMalloc(&a_d,sizeof(float)*(numvar)*(numvar+1));
    hipMalloc(&b_d,sizeof(float)*(numvar)*(numvar+1));
    //Copying data to device from host
    hipMemcpy(a_d, temp_h, sizeof(float)*numvar*(numvar+1),hipMemcpyHostToDevice);
    //Defining size of Thread Block
    dim3 dimBlock(numvar+1,numvar,1);
    dim3 dimGrid(1,1,1);
    //Kernel call 
    //Kernel<<<dimGrid , dimBlock>>>(a_d , b_d , numvar);
    //Coping data to host from device
    hipMemcpy(temp1_h,b_d,sizeof(float)*numvar*(numvar+1),hipMemcpyDeviceToHost);
    //Deallocating memory on the device
    hipFree(a_d);
    hipFree(b_d);
}

 __global__ void detMatrixOnGPUMix(float *matrix){
    int matrix_size, matrix_number, current_matrix;
    float mult, deter, pivot;

    matrix_size = blockDim.x;
    matrix_number = gridDim.y;
    current_matrix = blockIdx.y;

    unsigned int idxCollumn = threadIdx.x;
    unsigned int idxCurrentMatrix = current_matrix * matrix_size * matrix_size;
    unsigned int idxPivot = idxCurrentMatrix + idxCollumn * matrix_size + idxCollumn;

    pivot = matrix[idxPivot];
    //printf("%d %d %d %d %f\n", current_matrix, idxCollumn, idxCurrentMatrix, idxPivot, pivot);
    printf("idxCurrentMatrix: %d current_matrix: %d\n", idxCurrentMatrix, current_matrix);

    for (int i = idxCurrentMatrix; i < (idxCurrentMatrix+1) * matrix_size * matrix_size; i += matrix_size) {
        //if (current_matrix == 0)
            printf("%d %d %d %d\n", current_matrix, i, idxCollumn, i * matrix_size + idxCollumn);
    }

    if(pivot == 0){
	//to do - switch columns
	}

    /* Determinant Calculation */
    // Gauss Elimination
    for(int k = 0; k < matrix_size-1; k++) {
        for(int i = k+1; i < matrix_size; i++) {//por que k+1?
            //mult = matrix[i][k]/matrix[k][k];
            matrix[k*(i+1)] = matrix[1*(i+1)] * matrix[k*1] - matrix[k*(i+1)] * matrix[1*1];//esse código é necessário?
            __syncthreads();

            //for(int j = k+1; j <= matrix_size; j++){//por que k+1?
            //    matrix[i][j] -= mult * matrix[k][j];
		//__syncthread();
	    //}	
        }
    }

    // determinant calculation
    deter = 1;
    for(int i = 0; i < matrix_size; i++)
        deter += blockIdx.y;
        //deter *= matrix_calc[i][i];
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 31; i < 64; i++)
    {
        printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            //printf("%d host %f gpu %f\n", i, hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

int main(int argc, char **argv)
{
    char *fName;
	
	if (argc != 2){
		printf("Usage: ./<ExecutableFilename> <MatrixFilename>\n");
		printf("%6s MatrixFilename: Name the file that contains the matrices\n", "->");
		exit(1);
	}
	
	fName = argv[1];
	
	printf("%s Starting...\n", argv[0]);

	int matrix_number, current_matrix, matrixreceived = 0;
	FILE *matrix_file = fopen(fName, "rb");
	
    if(matrix_file == NULL) {
        printf("Cannot open the %s file\n"
               "Please check if is a valid matrix file and the user read permissions!\n", argv[1]);
        return 1;
    }
	
    fread(&matrix_number, sizeof(int), 1, matrix_file);
    fread(&matrix_size, sizeof(int), 1, matrix_file);
    int dimension  = matrix_number * matrix_size;

    printf("Matrix size: %d | Matrix count: %d\n", matrix_size, matrix_number);
    
    //Read the matrices on file 
    matrix = (float*) malloc(sizeof(float) * (matrix_size * matrix_size * matrix_number));
    for(current_matrix = 0; current_matrix < matrix_number; current_matrix++){
        for(int i = current_matrix * matrix_size * matrix_size; i < (current_matrix + 1) * matrix_size * matrix_size; i++) {
            fread(&matrix[i], sizeof(float), 1, matrix_file);
            //printf("%d %f\n", i, matrix[i]);
        }
    }
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = matrix_size;
    int ny = matrix_number;

    int nxy = nx * nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    //hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    double iStart = seconds();

    double iElaps = seconds() - iStart;
    printf("Matrix initialization elapsed %f sec\n", iElaps);

    //memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    iStart = seconds();

    //Multiply Matrix OnHost(h_A, h_B, hostRef, nx, ny);
    //multMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    iElaps = seconds() - iStart;
    
    //printf("multMatrixOnHost elapsed %f sec\n", iElaps);

    CHECK(hipMalloc((void **)&dev_matrix, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(dev_matrix, matrix, nBytes, hipMemcpyHostToDevice));
    
    dim3 grid(1,ny);
    dim3 block(nx,1);

    iStart = seconds();
    detMatrixOnGPUMix<<<grid, block>>>(dev_matrix);
    
    //CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("detMatrixOnGPUMix <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x, grid.y, block.x, block.y, iElaps);

    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    //CHECK(hipMemcpy(gpuRef, dev_matrix, nBytes, hipMemcpyDeviceToHost));

    // check device results
    //checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(dev_matrix));

    // free host memory
    free(matrix);
    //free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}
